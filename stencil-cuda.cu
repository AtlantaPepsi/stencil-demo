#include "hip/hip_runtime.h"
#include "prk_util.h"
#include "prk_cuda.h"

__global__ void star2(const int n, const double * in, double * out) {
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ( (2 <= i) && (i < n-2) && (2 <= j) && (j < n-2) ) {
            out[i*n+j] += +in[(i)*n+(j-2)] * -0.125
                          +in[(i)*n+(j-1)] * -0.25
                          +in[(i-2)*n+(j)] * -0.125
                          +in[(i-1)*n+(j)] * -0.25
                          +in[(i+1)*n+(j)] * 0.25
                          +in[(i+2)*n+(j)] * 0.125
                          +in[(i)*n+(j+1)] * 0.25
                          +in[(i)*n+(j+2)] * 0.125;
     }
}

__global__ void star3(const int n, const double * in, double * out) {
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ( (3 <= i) && (i < n-3) && (3 <= j) && (j < n-3) ) {
            out[i*n+j] += +in[(i)*n+(j-3)] * -0.05555555555555555
                          +in[(i)*n+(j-2)] * -0.08333333333333333
                          +in[(i)*n+(j-1)] * -0.16666666666666666
                          +in[(i-3)*n+(j)] * -0.05555555555555555
                          +in[(i-2)*n+(j)] * -0.08333333333333333
                          +in[(i-1)*n+(j)] * -0.16666666666666666
                          +in[(i+1)*n+(j)] * 0.16666666666666666
                          +in[(i+2)*n+(j)] * 0.08333333333333333
                          +in[(i+3)*n+(j)] * 0.05555555555555555
                          +in[(i)*n+(j+1)] * 0.16666666666666666
                          +in[(i)*n+(j+2)] * 0.08333333333333333
                          +in[(i)*n+(j+3)] * 0.05555555555555555;
     }
}

__global__ void star4(const int n, const double * in, double * out) {
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    if ( (4 <= i) && (i < n-4) && (4 <= j) && (j < n-4) ) {
            out[i*n+j] += +in[(i)*n+(j-4)] * -0.03125
                          +in[(i)*n+(j-3)] * -0.041666666666666664
                          +in[(i)*n+(j-2)] * -0.0625
                          +in[(i)*n+(j-1)] * -0.125
                          +in[(i-4)*n+(j)] * -0.03125
                          +in[(i-3)*n+(j)] * -0.041666666666666664
                          +in[(i-2)*n+(j)] * -0.0625
                          +in[(i-1)*n+(j)] * -0.125
                          +in[(i+1)*n+(j)] * 0.125
                          +in[(i+2)*n+(j)] * 0.0625
                          +in[(i+3)*n+(j)] * 0.041666666666666664
                          +in[(i+4)*n+(j)] * 0.03125
                          +in[(i)*n+(j+1)] * 0.125
                          +in[(i)*n+(j+2)] * 0.0625
                          +in[(i)*n+(j+3)] * 0.041666666666666664
                          +in[(i)*n+(j+4)] * 0.03125;
     }
}

__global__ void nothing(const int n, const double * in, double * out)
{
}

__global__ void add(const int n, double * in)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i<n) && (j<n)) {
        in[i*n+j] += 1.0;
    }
}

int main(int argc, char* argv[])
{
  std::cout << "Parallel Research Kernels version " << std::endl;
  std::cout << "C++11/CUDA Stencil execution on 2D grid" << std::endl;

  //////////////////////////////////////////////////////////////////////
  // Process and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations;
  size_t n, block_size = 16, radius = 2;

  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <array dimension> [<block size> <stencil radius>]";
      }

      // number of times to run the algorithm
      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      // linear grid dimension
      n  = std::atoi(argv[2]);
      if (n < 1) {
        throw "ERROR: grid dimension must be positive";
      } else if (n > prk::get_max_matrix_size()) {
        throw "ERROR: grid dimension too large - overflow risk";
      }

      if (argc > 3) {
          block_size = std::atoi(argv[3]);
          if (block_size <= 0) block_size = n;
          if (block_size > n) block_size = n;
      }
      if (n % block_size) {
        throw "ERROR: block size does not evenly divide grid size";
      }

      // stencil radius
      radius = 2;
      if (argc > 4) {
          radius = std::atoi(argv[4]);
      }

      if ( (radius < 1) || (2*radius+1 > n) ) {
        throw "ERROR: Stencil radius negative or too large";
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Grid size            = " << n << std::endl;
  std::cout << "Block size           = " << block_size << std::endl;
  std::cout << "Radius of stencil    = " << radius << std::endl;

  //////////////////////////////////////////////////////////////////////
  /// Setup CUDA environment
  //////////////////////////////////////////////////////////////////////

  prk::CUDA::info info;
  info.print(1);

  auto stencil = nothing;
  switch (radius) {
      case 2: stencil = star2; break;
      case 3: stencil = star3; break;
      case 4: stencil = star4; break;
  }

  dim3 dimGrid(prk::divceil(n,block_size),prk::divceil(n,block_size),1);
  dim3 dimBlock(block_size, block_size, 1);
  info.checkDims(dimBlock, dimGrid);

  //////////////////////////////////////////////////////////////////////
  // Allocate space and perform the computation
  //////////////////////////////////////////////////////////////////////

  double stencil_time{0};

  const size_t nelems = n*n;
  const size_t bytes = nelems * sizeof(double);
  double * h_in;
  double * h_out;
  prk::CUDA::check( hipHostMalloc((void**)&h_in, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_out, bytes) );

  for (int i=0; i<n; i++) {
    for (int j=0; j<n; j++) {
      h_in[i*n+j]  = static_cast<double>(i+j);
      h_out[i*n+j] = static_cast<double>(0);
    }
  }

  // copy input from host to device
  double * d_in;
  double * d_out;
  prk::CUDA::check( hipMalloc((void**)&d_in, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_out, bytes) );
  prk::CUDA::check( hipMemcpy(d_in, &(h_in[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_out, &(h_out[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipDeviceSynchronize() );

  for (int iter = 0; iter<=iterations; iter++) {

    if (iter==1) stencil_time = prk::wtime();

    // Apply the stencil operator
    stencil<<<dimGrid, dimBlock>>>(n, d_in, d_out);

    // Add constant to solution to force refresh of neighbor data, if any
    add<<<dimGrid, dimBlock>>>(n, d_in);

    prk::CUDA::check( hipDeviceSynchronize() );
  }
  stencil_time = prk::wtime() - stencil_time;

  // copy output back to host
  prk::CUDA::check( hipMemcpy(&(h_out[0]), d_out, bytes, hipMemcpyDeviceToHost) );

#ifdef VERBOSE
  // copy input back to host - debug only
  prk::CUDA::check( hipMemcpy(&(h_in[0]), d_in, bytes, hipMemcpyDeviceToHost) );
#endif

  prk::CUDA::check( hipFree(d_out) );
  prk::CUDA::check( hipFree(d_in) );

  //////////////////////////////////////////////////////////////////////
  // Analyze and output results
  //////////////////////////////////////////////////////////////////////

  // interior of grid with respect to stencil
  const size_t active_points = (n-2L*radius)*(n-2L*radius);
  double norm{0};
  for (size_t i=radius; i<n-radius; i++) {
    for (size_t j=radius; j<n-radius; j++) {
      norm += prk::abs(h_out[i*n+j]);
    }
  }
  norm /= active_points;

  // verify correctness
  const double epsilon = 1.0e-8;
  const double reference_norm = 2*(iterations+1);
  if (prk::abs(norm-reference_norm) > epsilon) {
    std::cout << "ERROR: L1 norm = " << norm
              << " Reference L1 norm = " << reference_norm << std::endl;
    return 1;
  } else {
    std::cout << "Solution validates" << std::endl;
#ifdef VERBOSE
    std::cout << "L1 norm = " << norm
              << " Reference L1 norm = " << reference_norm << std::endl;
#endif
    const size_t stencil_size = 4*radius+1;
    size_t flops = (2L*stencil_size+1L) * active_points;
    double avgtime = stencil_time/iterations;
    std::cout << 8*sizeof(double) << "B "
              << "Rate (MFlops/s): " << 1.0e-6 * static_cast<double>(flops)/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  }

  return 0;
}
