#include "hip/hip_runtime.h"
#include "prk_util.h"
#include "prk_cuda.h"

__global__ void star2(const int n, const double * in, double * out) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ( (2 <= i) && (i < n-2) && (2 <= j) && (j < n-2) ) {
            out[i*n+j] += +in[(i)*n+(j-2)] * -0.125
                          +in[(i)*n+(j-1)] * -0.25
                          +in[(i-2)*n+(j)] * -0.125
                          +in[(i-1)*n+(j)] * -0.25
                          +in[(i+1)*n+(j)] * 0.25
                          +in[(i+2)*n+(j)] * 0.125
                          +in[(i)*n+(j+1)] * 0.25
                          +in[(i)*n+(j+2)] * 0.125;
     }
}

__global__ void star3(const int n, const double * in, double * out) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ( (3 <= i) && (i < n-3) && (3 <= j) && (j < n-3) ) {
            out[i*n+j] += +in[(i)*n+(j-3)] * -0.05555555555555555
                          +in[(i)*n+(j-2)] * -0.08333333333333333
                          +in[(i)*n+(j-1)] * -0.16666666666666666
                          +in[(i-3)*n+(j)] * -0.05555555555555555
                          +in[(i-2)*n+(j)] * -0.08333333333333333
                          +in[(i-1)*n+(j)] * -0.16666666666666666
                          +in[(i+1)*n+(j)] * 0.16666666666666666
                          +in[(i+2)*n+(j)] * 0.08333333333333333
                          +in[(i+3)*n+(j)] * 0.05555555555555555
                          +in[(i)*n+(j+1)] * 0.16666666666666666
                          +in[(i)*n+(j+2)] * 0.08333333333333333
                          +in[(i)*n+(j+3)] * 0.05555555555555555;
     }
}

__global__ void star4(const int n, const double * in, double * out) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    if ( (4 <= i) && (i < n-4) && (4 <= j) && (j < n-4) ) {
            out[i*n+j] += +in[(i)*n+(j-4)] * -0.03125
                          +in[(i)*n+(j-3)] * -0.041666666666666664
                          +in[(i)*n+(j-2)] * -0.0625
                          +in[(i)*n+(j-1)] * -0.125
                          +in[(i-4)*n+(j)] * -0.03125
                          +in[(i-3)*n+(j)] * -0.041666666666666664
                          +in[(i-2)*n+(j)] * -0.0625
                          +in[(i-1)*n+(j)] * -0.125
                          +in[(i+1)*n+(j)] * 0.125
                          +in[(i+2)*n+(j)] * 0.0625
                          +in[(i+3)*n+(j)] * 0.041666666666666664
                          +in[(i+4)*n+(j)] * 0.03125
                          +in[(i)*n+(j+1)] * 0.125
                          +in[(i)*n+(j+2)] * 0.0625
                          +in[(i)*n+(j+3)] * 0.041666666666666664
                          +in[(i)*n+(j+4)] * 0.03125;
     }
}

__global__ void nothing(const int n, const double * in, double * out)
{
}

__global__ void add(const int n, double * in)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if ((i<n) && (j<n)) {
        in[i*n+j] += (double)1;
    }
}

int main(int argc, char* argv[])
{
  std::cout << "Parallel Research Kernels version " << std::endl;
  std::cout << "C++11/CUDA Stencil execution on 2D grid" << std::endl;

  prk::CUDA::info info;
  //info.print();

  //////////////////////////////////////////////////////////////////////
  // Process and test input parameters
  //////////////////////////////////////////////////////////////////////

  int iterations, n, radius, tile_size;
  bool star = true;
  try {
      if (argc < 3) {
        throw "Usage: <# iterations> <array dimension> [<tile_size> <star/grid> <radius>]";
      }

      // number of times to run the algorithm
      iterations  = std::atoi(argv[1]);
      if (iterations < 1) {
        throw "ERROR: iterations must be >= 1";
      }

      // linear grid dimension
      n  = std::atoi(argv[2]);
      if (n < 1) {
        throw "ERROR: grid dimension must be positive";
      } else if (n > prk::get_max_matrix_size()) {
        throw "ERROR: grid dimension too large - overflow risk";
      }

      // default tile size for tiling of local transpose
      tile_size = 32;
      if (argc > 3) {
          tile_size = std::atoi(argv[3]);
          if (tile_size <= 0) tile_size = n;
          if (tile_size > n) tile_size = n;
          if (tile_size > 32) {
              std::cout << "Warning: tile_size > 32 may lead to incorrect results (observed for CUDA 9.0 on GV100).\n";
          }
      }

      // stencil pattern
      if (argc > 4) {
          auto stencil = std::string(argv[4]);
          auto grid = std::string("grid");
          star = (stencil == grid) ? false : true;
      }

      // stencil radius
      radius = 2;
      if (argc > 5) {
          radius = std::atoi(argv[5]);
      }

      if ( (radius < 1) || (2*radius+1 > n) ) {
        throw "ERROR: Stencil radius negative or too large";
      }
  }
  catch (const char * e) {
    std::cout << e << std::endl;
    return 1;
  }

  std::cout << "Number of iterations = " << iterations << std::endl;
  std::cout << "Grid size            = " << n << std::endl;
  std::cout << "Tile size            = " << tile_size << std::endl;
  std::cout << "Type of stencil      = " << (star ? "star" : "grid") << std::endl;
  std::cout << "Radius of stencil    = " << radius << std::endl;

  auto stencil = nothing;
  if (star) {
      switch (radius) {
          case 2: stencil = star2; break;
          case 3: stencil = star3; break;
          case 4: stencil = star4; break;
      }
  }

  dim3 dimGrid(prk::divceil(n,tile_size),prk::divceil(n,tile_size),1);
  dim3 dimBlock(tile_size, tile_size, 1);
  info.checkDims(dimBlock, dimGrid);

  //////////////////////////////////////////////////////////////////////
  // Allocate space and perform the computation
  //////////////////////////////////////////////////////////////////////

  double stencil_time{0};

  const size_t nelems = (size_t)n * (size_t)n;
  const size_t bytes = nelems * sizeof(double);
  double * h_in;
  double * h_out;
  prk::CUDA::check( hipHostMalloc((void**)&h_in, bytes) );
  prk::CUDA::check( hipHostMalloc((void**)&h_out, bytes) );

  for (int i=0; i<n; i++) {
    for (int j=0; j<n; j++) {
      h_in[i*n+j]  = static_cast<double>(i+j);
      h_out[i*n+j] = static_cast<double>(0);
    }
  }

  // copy input from host to device
  double * d_in;
  double * d_out;
  prk::CUDA::check( hipMalloc((void**)&d_in, bytes) );
  prk::CUDA::check( hipMalloc((void**)&d_out, bytes) );
  prk::CUDA::check( hipMemcpy(d_in, &(h_in[0]), bytes, hipMemcpyHostToDevice) );
  prk::CUDA::check( hipMemcpy(d_out, &(h_out[0]), bytes, hipMemcpyHostToDevice) );

  for (int iter = 0; iter<=iterations; iter++) {

    if (iter==1) stencil_time = prk::wtime();

    // Apply the stencil operator
    stencil<<<dimGrid, dimBlock>>>(n, d_in, d_out);

    // Add constant to solution to force refresh of neighbor data, if any
    add<<<dimGrid, dimBlock>>>(n, d_in);

    prk::CUDA::check( hipDeviceSynchronize() );
  }
  stencil_time = prk::wtime() - stencil_time;

  // copy output back to host
  prk::CUDA::check( hipMemcpy(&(h_out[0]), d_out, bytes, hipMemcpyDeviceToHost) );

#ifdef VERBOSE
  // copy input back to host - debug only
  prk::CUDA::check( hipMemcpy(&(h_in[0]), d_in, bytes, hipMemcpyDeviceToHost) );
#endif

  prk::CUDA::check( hipFree(d_out) );
  prk::CUDA::check( hipFree(d_in) );

  //////////////////////////////////////////////////////////////////////
  // Analyze and output results.
  //////////////////////////////////////////////////////////////////////

  // interior of grid with respect to stencil
  size_t active_points = static_cast<size_t>(n-2*radius)*static_cast<size_t>(n-2*radius);
  double norm = 0.0;
  for (int i=radius; i<n-radius; i++) {
    for (int j=radius; j<n-radius; j++) {
      norm += prk::abs(h_out[i*n+j]);
    }
  }
  norm /= active_points;

  // verify correctness
  const double epsilon = 1.0e-8;
  double reference_norm = 2.*(iterations+1.);
  if (prk::abs(norm-reference_norm) > epsilon) {
    std::cout << "ERROR: L1 norm = " << norm
              << " Reference L1 norm = " << reference_norm << std::endl;
    return 1;
  } else {
    std::cout << "Solution validates" << std::endl;
#ifdef VERBOSE
    std::cout << "L1 norm = " << norm
              << " Reference L1 norm = " << reference_norm << std::endl;
#endif
    const int stencil_size = star ? 4*radius+1 : (2*radius+1)*(2*radius+1);
    size_t flops = (2L*(size_t)stencil_size+1L) * active_points;
    auto avgtime = stencil_time/iterations;
    std::cout << "Rate (MFlops/s): " << 1.0e-6 * static_cast<double>(flops)/avgtime
              << " Avg time (s): " << avgtime << std::endl;
  }

  return 0;
}
